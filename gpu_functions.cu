#include "libs.h"

void gpu_mode(int batch_size)
{
    hipdnnHandle_t cudnn_handle;
    hipEvent_t start, stop; // CUDA TIMING
    float time;
    std::vector<cv::Mat> images;
    int channels = 3;

    // Creating timing events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Loading images
    std::cout << "Loading images...\n";
    load_images(&images, batch_size);
    int image_rows = images.at(0).rows;
    int image_cols = images.at(0).cols;
    checkCUDNN(hipdnnCreate(&cudnn_handle));
    std::cout << "Preparing descriptors...\n";
    // Initializing descriptors
    hipdnnTensorDescriptor_t input_tensor;
    hipdnnTensorDescriptor_t output_tensor;
    hipdnnFilterDescriptor_t kernel_descriptor;
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    initializing_descriptors(&input_tensor, &output_tensor, &kernel_descriptor,
                             &convolution_descriptor, batch_size, channels, image_rows, image_cols);

    // Defining the algorithm
    hipdnnConvolutionFwdAlgo_t convolution_algorithm = defining_algorithm(cudnn_handle, input_tensor, output_tensor, kernel_descriptor, convolution_descriptor);

    // Allocating memory
    size_t workspace_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn_handle,
                                                       input_tensor,
                                                       kernel_descriptor,
                                                       convolution_descriptor,
                                                       output_tensor,
                                                       convolution_algorithm,
                                                       &workspace_bytes));

    void *d_workspace;     // Device workspace memory
    void *d_input_tensor;  // Device input tensor memory
    void *d_output_tensor; // Device output tensor memory
    void *d_kernel;        // Device kernel memory
    int tensor_size = batch_size * channels * image_rows * image_cols * sizeof(float);
    std::cout << "Allocating device memory...\n";

    // Start timer.
    std::cout << "Timer starts now.\n";
    hipEventRecord(start, 0);

    hipMalloc(&d_workspace, workspace_bytes);
    checkCUDAError("hipMalloc");
    allocate_memory(images, &d_input_tensor, &d_output_tensor, &d_kernel, tensor_size);

    // Performing algorithm
    std::cout << "Performing the convolution...\n";
    const float alpha = 1, beta = 0;
    hipdnnConvolutionForward(cudnn_handle,
                            &alpha,
                            input_tensor,
                            d_input_tensor,
                            kernel_descriptor,
                            d_kernel,
                            convolution_descriptor,
                            convolution_algorithm,
                            d_workspace,
                            workspace_bytes,
                            &beta,
                            output_tensor,
                            d_output_tensor);

    // Moving result back to CPU
    float *h_output_tensor;
    h_output_tensor = (float *)malloc(batch_size * channels * image_rows * image_cols * sizeof(float));
    hipMemcpy(h_output_tensor, d_output_tensor, tensor_size, hipMemcpyDeviceToHost);

    // Stop timer.
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Estimating time difference between start and stop, then storing the result in time.
    hipEventElapsedTime(&time, start, stop);

    // Saving images
    save_images(h_output_tensor, image_rows, image_cols, batch_size);

    printf("Execution time for convolution: %8.2f milliseconds\n", time);
    std::cout << "Results can be found in the outputs/gpu/ folder.\n";

    // Cleaning memory
    hipdnnDestroyTensorDescriptor(input_tensor);
    hipdnnDestroyTensorDescriptor(output_tensor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
    hipFree(d_workspace);
    hipFree(d_input_tensor);
    hipFree(d_output_tensor);
    hipFree(d_kernel);
    free(h_output_tensor);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipdnnDestroy(cudnn_handle);
}

void initializing_descriptors(hipdnnTensorDescriptor_t *input_tensor,
                              hipdnnTensorDescriptor_t *output_tensor,
                              hipdnnFilterDescriptor_t *kernel_descriptor,
                              hipdnnConvolutionDescriptor_t *convolution_descriptor,
                              int batch_size,
                              int channels,
                              int image_rows,
                              int image_cols)
{

    // Input tensor descriptor
    checkCUDNN(hipdnnCreateTensorDescriptor(input_tensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(*input_tensor,
                                          HIPDNN_TENSOR_NHWC, // format
                                          HIPDNN_DATA_FLOAT,  // data type
                                          batch_size,        // batch size
                                          channels,          // channels
                                          image_rows,        // height
                                          image_cols));      // width
    // Output tensor descriptor
    checkCUDNN(hipdnnCreateTensorDescriptor(output_tensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(*output_tensor,
                                          HIPDNN_TENSOR_NHWC, // format
                                          HIPDNN_DATA_FLOAT,  // data type
                                          batch_size,        // batch size
                                          channels,          // channels
                                          image_rows,        // height
                                          image_cols));      // width

    // Kernel descriptor
    checkCUDNN(hipdnnCreateFilterDescriptor(kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(*kernel_descriptor,
                                          HIPDNN_DATA_FLOAT,  // data type
                                          HIPDNN_TENSOR_NCHW, // format
                                          channels,          // out channels
                                          channels,          // in channels
                                          3,                 // height
                                          3));               // width

    // Convolution descriptor
    checkCUDNN(hipdnnCreateConvolutionDescriptor(convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(*convolution_descriptor,
                                               1,                       // pad height
                                               1,                       // pad width
                                               1,                       // vertical_stride
                                               1,                       // horizontal_stride
                                               1,                       // dilation_height
                                               1,                       // dilation_width
                                               HIPDNN_CROSS_CORRELATION, // mode
                                               HIPDNN_DATA_FLOAT));      // computeType
}

hipdnnConvolutionFwdAlgo_t defining_algorithm(hipdnnHandle_t cudnn_handle,
                                             hipdnnTensorDescriptor_t input_tensor,
                                             hipdnnTensorDescriptor_t output_tensor,
                                             hipdnnFilterDescriptor_t kernel_descriptor,
                                             hipdnnConvolutionDescriptor_t convolution_descriptor){
    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    // Convolution algorithm
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn_handle,
                                                   input_tensor,
                                                   kernel_descriptor,
                                                   convolution_descriptor,
                                                   output_tensor,
                                                   HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                   0, // Memory limit in bytes
                                                   &convolution_algorithm));
    return convolution_algorithm;
}

void allocate_memory(std::vector<cv::Mat> images, void **d_input_tensor,
                        void **d_output_tensor, void **d_kernel, int tensor_size)
{
    // Allocating device input tensor memory
    hipMalloc(d_input_tensor, tensor_size);
    checkCUDAError("hipMalloc");
    int rows = images.at(0).rows;
    int cols = images.at(0).cols;
    for (int i = 0; i < images.size(); i++)
    {
        size_t offset = i * 3 * rows * cols * sizeof(float);
        hipMemcpy(static_cast<char *>(*d_input_tensor) + offset,
                   images[i].ptr<float>(0), 3 * rows * cols * sizeof(float),
                   hipMemcpyHostToDevice);
        checkCUDAError("hipMemcpy");
    }
    // Allocating device output tensor memory
    hipMalloc(d_output_tensor, tensor_size);
    checkCUDAError("hipMalloc");
    hipMemset(*d_output_tensor, 0, tensor_size);
    checkCUDAError("hipMemset");
    // Allocating device kernel memory
    float h_kernel[3][3][3][3];
    for (int kernel = 0; kernel < 3; ++kernel){
        for (int channel = 0; channel < 3; ++channel){
            for (int row = 0; row < 3; ++row){
                for (int column = 0; column < 3; ++column){
                    h_kernel[kernel][channel][row][column] = kernel_template[row][column];
                }
            }
        }
    }
    hipMalloc(d_kernel, sizeof(h_kernel));
    checkCUDAError("hipMalloc");
    hipMemcpy(*d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy");
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        // hipGetErrorString(err) prende l'errore in input, che e' di tipo cudastranoerror, e lo trasforma in una stringa stampabile a schermo.
        fprintf(stderr, "Cuda error: %s %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void checkCUDNN(hipdnnStatus_t expression)
{
    hipdnnStatus_t status = (expression);
    if (status != HIPDNN_STATUS_SUCCESS)
    {
        std::cerr << "Error on line " << __LINE__ << ": "
                  << hipdnnGetErrorString(status) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}